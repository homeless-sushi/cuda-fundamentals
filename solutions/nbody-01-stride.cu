#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "files.h"

#define SOFTENING 1e-9f

typedef struct { float x, y, z, vx, vy, vz; } Body;

// A simple parallelization of the outer for block.
// We use every thread for multiple data points
__global__
void bodyForce(Body *p, float dt, int n) {
  unsigned int absoluteThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = absoluteThreadIndex; i < n; i+=stride){
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
        for (int j = 0; j < n; j++) {
        float dx = p[j].x - p[i].x;
        float dy = p[j].y - p[i].y;
        float dz = p[j].z - p[i].z;
        float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
        float invDist = rsqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;

        Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
        }

        p[i].vx += dt*Fx; 
        p[i].vy += dt*Fy; 
        p[i].vz += dt*Fz;
  }
}

int main(const int argc, const char** argv) {

  // The assessment will test against both 2<11 and 2<15.
  // Feel free to pass the command line argument 15 when you generate ./nbody report files
  int nBodies = 2<<11;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  // The assessment will pass hidden initialized values to check for correctness.
  // You should not make changes to these files, or else the assessment will not work.
  const char * initialized_values;
  const char * solution_values;

  if (nBodies == 2<<11) {
    initialized_values = "09-nbody/files/initialized_4096";
    solution_values = "09-nbody/files/solution_4096";
  } else { // nBodies == 2<<15
    initialized_values = "09-nbody/files/initialized_65536";
    solution_values = "09-nbody/files/solution_65536";
  }

  if (argc > 2) initialized_values = argv[2];
  if (argc > 3) solution_values = argv[3];

  const float dt = 0.01f; // Time step
  int nIters = 10;  // Simulation iterations

  int bytes = nBodies * sizeof(Body);
  float *buf;
  buf = (float *)malloc(bytes);
  Body *p = (Body*)buf;

  Body* p_d;
  hipMalloc(&p_d, bytes);

  int device;
  hipGetDevice(&device);
  hipDeviceProp_t device_properties;
  hipGetDeviceProperties(&device_properties, device);

  read_values_from_file(initialized_values, buf, bytes);

  const unsigned int blockDim_x = 64;
  const unsigned int max_blocks 
    = max(device_properties.multiProcessorCount*8, (nBodies + blockDim_x - 1) / blockDim_x);
  
  for (int iter = 0; iter < nIters; iter++) {

    hipMemcpy(p_d, p, bytes, hipMemcpyHostToDevice);
    bodyForce<<<max_blocks, blockDim_x>>>(p_d, dt, nBodies); 
    hipMemcpy(p, p_d, bytes, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    for (int i = 0 ; i < nBodies; i++) { 
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }
  }

  write_values_to_file(solution_values, buf, bytes);
  
  free(buf);
  hipDeviceReset();
}
