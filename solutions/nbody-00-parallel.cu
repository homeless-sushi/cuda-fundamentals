#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "files.h"

#define SOFTENING 1e-9f

typedef struct { float x, y, z, vx, vy, vz; } Body;

// A simple parallelization of the outer for block.
// We create a thread for every data point
__global__
void bodyForce(Body *p, float dt, int n) {
  unsigned int absoluteThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
  
  float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[absoluteThreadIndex].x;
      float dy = p[j].y - p[absoluteThreadIndex].y;
      float dz = p[j].z - p[absoluteThreadIndex].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[absoluteThreadIndex].vx += dt*Fx; 
    p[absoluteThreadIndex].vy += dt*Fy; 
    p[absoluteThreadIndex].vz += dt*Fz;
}

int main(const int argc, const char** argv) {

  // The assessment will test against both 2<11 and 2<15.
  // Feel free to pass the command line argument 15 when you generate ./nbody report files
  int nBodies = 2<<11;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  // The assessment will pass hidden initialized values to check for correctness.
  // You should not make changes to these files, or else the assessment will not work.
  const char * initialized_values;
  const char * solution_values;

  if (nBodies == 2<<11) {
    initialized_values = "09-nbody/files/initialized_4096";
    solution_values = "09-nbody/files/solution_4096";
  } else { // nBodies == 2<<15
    initialized_values = "09-nbody/files/initialized_65536";
    solution_values = "09-nbody/files/solution_65536";
  }

  if (argc > 2) initialized_values = argv[2];
  if (argc > 3) solution_values = argv[3];

  const float dt = 0.01f; // Time step
  int nIters = 10;  // Simulation iterations

  int bytes = nBodies * sizeof(Body);
  float *buf;
  buf = (float *)malloc(bytes);
  Body *p = (Body*)buf;

  Body* p_d;
  hipMalloc(&p_d, bytes);

  read_values_from_file(initialized_values, buf, bytes);

  const unsigned int blockDim_x = 64;
  const unsigned int max_blocks = (nBodies + blockDim_x - 1) / blockDim_x;
  
  for (int iter = 0; iter < nIters; iter++) {

    hipMemcpy(p_d, p, bytes, hipMemcpyHostToDevice);
    bodyForce<<<max_blocks, blockDim_x>>>(p_d, dt, nBodies); 
    hipMemcpy(p, p_d, bytes, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    for (int i = 0 ; i < nBodies; i++) { 
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }
  }

  write_values_to_file(solution_values, buf, bytes);

  free(buf);
  hipDeviceReset();
}
